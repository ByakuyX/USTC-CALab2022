
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
#define BLOCK_SIZE 32
#define BS 32

int N;

__device__ float* GetSubMatrix(float *A, int row, int col, int N)
{
    float *Asub = A + (BS * row) * N + (BS * col);
    return Asub;
}
__device__ void SetElement(float *A, int row, int col, float val, int N)
{
    A[row * N + col] = val; 
}
__device__ float GetElement(float *A,int row, int col, int N)
{
    return A[row * N + col];
}

__global__ void gemm_baseline(float *A, float *B, float *C, int N)
{
	int block_row = blockIdx.y;
    int block_col = blockIdx.x;
    float *Csub = GetSubMatrix(C, block_row, block_col, N);
    float Cval = 0;
    int row = threadIdx.y;
    int col = threadIdx.x;
    for(int i = 0; i < N / BS; i ++)
    {
        float *Asub = GetSubMatrix(A, block_row, i, N);
        float *Bsub = GetSubMatrix(B, i, block_col, N);
        __shared__ float As[BS][BS];
        __shared__ float Bs[BS][BS];
        As[row][col] = GetElement(Asub, row, col, N);
        Bs[row][col] = GetElement(Bsub, row, col, N);
        __syncthreads();
        for(int e = 0; e < BS; e ++)
            Cval += As[row][e] * Bs[e][col];
        __syncthreads();
    }
    SetElement(Csub, row, col, Cval, N);
}

void gemm_verify(float *A, float *B, float *C)
{
    size_t size = N * N * sizeof(float);
    float *DA;
    float *DB;
    float *DC;
    hipMalloc(&DA, size);
    hipMalloc(&DB, size);
    hipMalloc(&DC, size);
    hipMemcpy(DA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(DB, B, size, hipMemcpyHostToDevice);
    dim3 dimBl(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGr((N + dimBl.x - 1) / dimBl.x, (N + dimBl.y - 1) / dimBl.y);
    gemm_baseline<<<dimBl, dimGr>>>(DA, DB, DC, N);
	hipMemcpy(C, DC, size, hipMemcpyDeviceToHost);
    hipFree(DA);
    hipFree(DB);
    hipFree(DC);
}

int main()
{
    scanf("%d", &N);
    N = (1 << N);
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));
    for(int i = 0; i < N * N; i ++)
    {
        A[i] = rand() / (double)RAND_MAX;
        B[i] = rand() / (double)RAND_MAX;
        C[i] = 0;
    }
    gemm_verify(A, B, C);
    return 0;
}
