
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
#define BLOCK_SIZE 8

int N;

__global__ void gemm_baseline(float *A, float *B, float *C, int N)
{
    float c0 = 0;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N && j < N)
    {
        for(int k = 0; k < N; k ++)
            c0 += A[i * N + k] * B[k * N + j];
        C[i * N + j] = c0;
    }
}

void gemm_verify(float *A, float *B, float *C)
{
    size_t size = N * N * sizeof(float);
    float *DA;
    float *DB;
    float *DC;
    hipMalloc(&DA, size);
    hipMalloc(&DB, size);
    hipMalloc(&DC, size);
    hipMemcpy(DA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(DB, B, size, hipMemcpyHostToDevice);
    dim3 dimBl(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGr((N + dimBl.x - 1) / dimBl.x, (N + dimBl.y - 1) / dimBl.y);
    gemm_baseline<<<dimBl, dimGr>>>(DA, DB, DC, N);
    hipMemcpy(C, DC, size, hipMemcpyDeviceToHost);
    hipFree(DA);
    hipFree(DB);
    hipFree(DC);
}

int main()
{
    scanf("%d", &N);
    N = (1 << N);
    float *A = (float *)malloc(N * N * sizeof(float));
    float *B = (float *)malloc(N * N * sizeof(float));
    float *C = (float *)malloc(N * N * sizeof(float));
    for(int i = 0; i < N * N; i ++)
    {
        A[i] = rand() / (double)RAND_MAX;
        B[i] = rand() / (double)RAND_MAX;
        C[i] = 0;
    }
    gemm_verify(A, B, C);
    return 0;
}
